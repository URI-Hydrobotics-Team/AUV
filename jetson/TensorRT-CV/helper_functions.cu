#include "hip/hip_runtime.h"
#include "helper_functions.cuh"

__global__ void convert_to_float(int n, __half* FP16_ptr, float* FP32_ptr){
    // get the thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure the index does not go out of bounds
    if(i < n){
        // get the current FP16 value
        __half FP16_val = FP16_ptr[i];
        // convert it to FP32 value
        float FP32_val = __half2float(FP16_val);
        // store it in the FP32_ptr
        FP32_ptr[i] = FP32_val;
    }
}

// launch function
namespace helper{
    void convert(int n, __half* FP16_ptr, float* FP32_ptr){
        int num_threads = 512;
        int num_block = (n + num_threads - 1) / num_threads;
        convert_to_float<<<num_block, num_threads>>>(n, FP16_ptr, FP32_ptr);
    }
}